
#include <hip/hip_runtime.h>
#define IN_RANGE(x, min, max)   ((x)>=(min) && (x)<=(max))
#define BLOCK_SIZE_REF 16 //has to be the value as in hotspot_reference.h

extern "C" __global__ void hotspot(  int iteration,
                                float *power,
                                float *temp_src,
                                float *temp_dst,
                               int grid_cols,
                               int grid_rows,
							   int border_cols,
							   int border_rows,
                               float Cap,
                               float Rx, 
                               float Ry, 
                               float Rz, 
                               float step) {
	
	__shared__ float temp_on_cuda[BLOCK_SIZE_REF][BLOCK_SIZE_REF];
	__shared__ float power_on_cuda[BLOCK_SIZE_REF][BLOCK_SIZE_REF];
	__shared__ float temp_t[BLOCK_SIZE_REF][BLOCK_SIZE_REF]; // saving temporary temperature result

	float amb_temp = 80.0f;
	float step_div_Cap;
	float Rx_1,Ry_1,Rz_1;

	int bx = blockIdx.x;
	int by = blockIdx.y;

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	step_div_Cap=step/Cap;

	Rx_1=1/Rx;
	Ry_1=1/Ry;
	Rz_1=1/Rz;

	// each block finally computes result for a small block
	// after N iterations. 
	// it is the non-overlapping small blocks that cover 
	// all the input data

	// calculate the small block size
	int small_block_rows = BLOCK_SIZE_REF-iteration*2;//EXPAND_RATE
	int small_block_cols = BLOCK_SIZE_REF-iteration*2;//EXPAND_RATE

	// calculate the boundary for the block according to 
	// the boundary of its small block
	int blkY = small_block_rows*by-border_rows;
	int blkX = small_block_cols*bx-border_cols;
	int blkYmax = blkY+BLOCK_SIZE_REF-1;
	int blkXmax = blkX+BLOCK_SIZE_REF-1;

	// calculate the  thread coordination
	int yidx = blkY+ty;
	int xidx = blkX+tx;

	// load data if it is within the valid input range
	int loadYidx=yidx, loadXidx=xidx;
	int index = grid_cols*loadYidx+loadXidx;
       
	if(IN_RANGE(loadYidx, 0, grid_rows-1) && IN_RANGE(loadXidx, 0, grid_cols-1)){
            temp_on_cuda[ty][tx] = temp_src[index];  // Load the temperature data from  memory to shared memory
            power_on_cuda[ty][tx] = power[index];// Load the power data from  memory to shared memory
	}
	__syncthreads();

	// effective range within this block that falls within 
	// the valid range of the input data
	// used to rule out computation outside the boundary.
	int validYmin = (blkY < 0) ? -blkY : 0;
	int validYmax = (blkYmax > grid_rows-1) ? BLOCK_SIZE_REF-1-(blkYmax-grid_rows+1) : BLOCK_SIZE_REF-1;
	int validXmin = (blkX < 0) ? -blkX : 0;
	int validXmax = (blkXmax > grid_cols-1) ? BLOCK_SIZE_REF-1-(blkXmax-grid_cols+1) : BLOCK_SIZE_REF-1;

	int N = ty-1;
	int S = ty+1;
	int W = tx-1;
	int E = tx+1;

	N = (N < validYmin) ? validYmin : N;
	S = (S > validYmax) ? validYmax : S;
	W = (W < validXmin) ? validXmin : W;
	E = (E > validXmax) ? validXmax : E;

	bool computed;
	for (int i=0; i<iteration ; i++){ 
		computed = false;
		if( IN_RANGE(tx, i+1, BLOCK_SIZE_REF-i-2) &&  \
		IN_RANGE(ty, i+1, BLOCK_SIZE_REF-i-2) &&  \
		IN_RANGE(tx, validXmin, validXmax) && \
		IN_RANGE(ty, validYmin, validYmax) ) {
			computed = true;
			temp_t[ty][tx] =   temp_on_cuda[ty][tx] + step_div_Cap * (power_on_cuda[ty][tx] + 
			(temp_on_cuda[S][tx] + temp_on_cuda[N][tx] - 2.0f * temp_on_cuda[ty][tx]) * Ry_1 + 
			(temp_on_cuda[ty][E] + temp_on_cuda[ty][W] - 2.0f * temp_on_cuda[ty][tx]) * Rx_1 + 
			(amb_temp - temp_on_cuda[ty][tx]) * Rz_1);
		}
		__syncthreads();
		
		if(i==iteration-1)
			break;
		if(computed)	 //Assign the computation range
			temp_on_cuda[ty][tx]= temp_t[ty][tx];
			
		__syncthreads();
	}

	// update the  memory
	// after the last iteration, only threads coordinated within the 
	// small block perform the calculation and switch on ``computed''
	if (computed){
	  temp_dst[index]= temp_t[ty][tx];		
	}
}
